#include "manager.cuh"
#include "load_model.h"
#include "../ops/elementwise.cuh"

global_handle::global_handle (bool BERT_Large, std::string dir) {
    if(BERT_Large){
        dir_npy = "model_npy/large_uncased";
        hidden_size = 1024;
        num_hidden_layers = 24;
        num_attention_heads = 16;
        intermediate_size = 4096;
        max_mem_size = 200 * 512;
    }
    if (dir != "") 
        dir_npy = dir;
    load_from_dir_to_GPU(dir_npy, tts);
    checkError(hipblasCreate(&handle), "hipblasCreate() error!\n");
    init_cudamemory(max_mem_size / max_seq_length, max_seq_length);
    hipStreamCreate(&cal_stream);
    hipStreamCreate(&copy_stream);
    hipEventCreate(&copy_event);
    checkError(hipblasSetStream(handle, cal_stream), "Set cublas stream Error!\n");
}

global_handle::~global_handle(){
    global_malloc_manage_float.del();
    global_malloc_manage_int.del();
    checkError(hipblasDestroy(handle), "hipblasDestroy() error!\n");
}

void global_handle::init_cudamemory(int batchsize, int seq_length){
    global_malloc_manage_int.init(batchsize * seq_length * 4);

    size_t left, total, real_Memcost;
    checkCudaErrors(hipMemGetInfo(&left, &total));
    left -= 1024 * 1024 * 500;
    std::cout<<"CUDA Memory INFO: Free: "<< left / 1024 / 1024 <<"MB"<<std::endl;
    left = left / sizeof(float);
    global_malloc_manage_float.init(left);
    
    while(1){
        real_Memcost =  batchsize*seq_length*hidden_size + 
                        batchsize*hidden_size*3 +  
                        1 * 
                        (batchsize*seq_length*hidden_size*6 + 
                        batchsize * num_attention_heads * seq_length*seq_length +
                        batchsize*hidden_size*seq_length*3 + 
                        batchsize*seq_length*2 + 
                        batchsize*seq_length * intermediate_size*2 + 
                        batchsize*seq_length*2 +
                        3*hidden_size*hidden_size) + 
                        batchsize*hidden_size*2 +
                        batchsize*seq_length*hidden_size;
        if (real_Memcost < left)
            break;
        else
            batchsize = batchsize * 9 / 10;
    }
    
    max_mem_size = batchsize * seq_length;
    std::cout<<"Support max_seq_length: "<<max_seq_length<<" max_batchsize: "
                <<batchsize<<" approximate max_size: "<<max_mem_size<<std::endl;
    
}

void global_handle::set_scale(size_t input_batchsize, size_t input_seq_length){
    batchsize = input_batchsize;
    seq_length = input_seq_length;
    if(batchsize * seq_length > max_mem_size){
        std::cout<<"Error : Batchsize * Seq_lengh is too big too alloc"<<std::endl;
        std::cout<<" batchsize: "<<batchsize<<" seq_length: "
                    <<seq_length<<" max_size: "<<max_mem_size<<std::endl;
        assert(batchsize * seq_length <= max_mem_size);
    }
}

