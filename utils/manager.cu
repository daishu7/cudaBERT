#include "manager.cuh"
#include "load_model.h"

global_handle::global_handle (bool BERT_Large, std::string dir, bool optimRunningTime, bool isTrain, int numClasses) {
    if(BERT_Large){
        dir_npy = "model_npy/large_uncased";
        hidden_size = 1024;
        num_hidden_layers = 24;
        num_attention_heads = 16;
        intermediate_size = 4096;
        max_mem_size = 200 * 512;
    }
    if (dir != "") 
        dir_npy = dir;
    optim_running_time = optimRunningTime;
    is_train = isTrain;
    num_classes = numClasses;
    load_from_dir_to_GPU(dir_npy, tts);
    checkError(hipblasCreate(&handle), "hipblasCreate() error!\n");
    init_cudamemory(max_mem_size / max_seq_length, max_seq_length);
    hipStreamCreate(&cal_stream);
    hipStreamCreate(&copy_stream);
    hipEventCreate(&copy_event);
    hipEventCreate(&layer_compute_done);
    checkError(hipblasSetStream(handle, cal_stream), "Set cublas stream Error!\n");
}

void global_handle::set_optim_sgd(float lr) {
    learning_rate = lr;
    optim_method = "sgd";
}

void global_handle::set_optim_adam(float lr, float weightDecayLate, float beta1,
                    float beta2, float eps) {
    learning_rate = lr;
    weight_decay_rate = weightDecayLate;
    beta_1 = beta1;
    beta_2 = beta2;
    epsilon = eps;
    optim_method = "adam";
}

void global_handle::set_optim_momentum(float lr, float beta) {
    learning_rate = lr;
    momentum_beta = beta;
}

global_handle::~global_handle(){
    global_malloc_manage_float.del();
    global_malloc_manage_int.del();
    checkError(hipblasDestroy(handle), "hipblasDestroy() error!\n");
}

void global_handle::init_cudamemory(int batchsize, int seq_length){
    global_malloc_manage_int.init(batchsize * seq_length * 4);

    size_t left, total, real_Memcost;
    checkCudaErrors(hipMemGetInfo(&left, &total));

    left -= 1024 * 1024 * 500;

    std::cout<<"CUDA Memory INFO: Free: "<< left / 1024 / 1024 <<"MB"<<std::endl;
    left = left / sizeof(float);
    global_malloc_manage_float.init(left);
    
    while(1){
        //TODO: train or inference
        real_Memcost =  batchsize*seq_length*hidden_size + 
                        batchsize*hidden_size*3 +  
                        1 * 
                        (batchsize*seq_length*hidden_size*6 + 
                        batchsize * num_attention_heads * seq_length*seq_length +
                        batchsize*hidden_size*seq_length*3 + 
                        batchsize*seq_length*2 + 
                        batchsize*seq_length * intermediate_size*2 + 
                        batchsize*seq_length*2 +
                        3*hidden_size*hidden_size) + 
                        batchsize*hidden_size*2 +
                        batchsize*seq_length*hidden_size;
        if (real_Memcost < left)
            break;
        else
            batchsize = batchsize * 9 / 10;
    }
    
    max_mem_size = batchsize * seq_length;
    std::cout<<"Support max_seq_length: "<<max_seq_length<<" max_batchsize: "
                <<batchsize<<" approximate max_size: "<<max_mem_size<<std::endl;
    
}

void global_handle::set_scale(size_t input_batchsize, size_t input_seq_length){
    batchsize = input_batchsize;
    seq_length = input_seq_length;
    if(batchsize * seq_length > max_mem_size){
        std::cout<<"Error : Batchsize * Seq_lengh is too big too alloc"<<std::endl;
        std::cout<<" batchsize: "<<batchsize<<" seq_length: "
                    <<seq_length<<" max_size: "<<max_mem_size<<std::endl;
        assert(batchsize * seq_length <= max_mem_size);
    }
}

