#include "hip/hip_runtime.h"
#include "bert.cuh"
#include "load_model.h"
#include "../ops/linear.cuh"

bert::bert(int max_batchsize,
           int max_seq_length,
           bool BERT_Large,
           int num_gpu, 
           std::string dir, 
           bool is_train, 
           bool optimRunningTime, 
           int num_classes,
           std::string optim_method, 
           float lr) {
    checkCudaErrors(hipSetDevice(num_gpu));
    handle = new global_handle(max_batchsize,
                               max_seq_length,
                               BERT_Large, 
                               dir, 
                               optimRunningTime, 
                               is_train, 
                               num_classes);
    if (is_train) {
        // TODO: optim 参数
        if (optim_method == "sgd")
            handle->set_optim_sgd(lr);
        else if (optim_method == "adam")
            handle->set_optim_adam(lr);
        else if (optim_method == "momentum")
            handle->set_optim_momentum(lr);
    }
    init_ops();
    handle->init_cudamemory(max_batchsize, max_seq_length);
}

void bert::init_ops() {

    for (int i = 0; i < handle->num_hidden_layers; i++) {
        std::string num_layer = "_" + std::to_string(i) + "_";

        op_LayerNorm *layernorm = new op_LayerNorm(num_layer + "attention_output_LayerNorm_gamma",
                                                   num_layer + "attention_output_LayerNorm_beta",
                                                   handle, handle->hidden_size);
        attention_layernorm.push_back(layernorm);

        layernorm = new op_LayerNorm(num_layer + "output_LayerNorm_gamma",
                                     num_layer + "output_LayerNorm_beta",
                                     handle, handle->hidden_size);
        output_layernorm.push_back(layernorm);

        op_SoftMax *Softmax = new op_SoftMax(handle);
        softmax.push_back(Softmax);

        op_Linear *linear = new op_Linear(num_layer + "attention_output_dense_kernel",
                                          num_layer + "attention_output_dense_bias",
                                          handle, handle->hidden_size * handle->hidden_size, 
                                          handle->hidden_size);
        attention_linear.push_back(linear);

        linear = new op_Linear(num_layer + "intermediate_dense_kernel",
                               num_layer + "intermediate_dense_bias",
                               handle, 
                               handle->hidden_size * handle->intermediate_size, 
                               handle->intermediate_size);
        intermediate_linear.push_back(linear);

        linear = new op_Linear(num_layer + "output_dense_kernel",
                               num_layer + "output_dense_bias",
                               handle, 
                               handle->intermediate_size * handle->hidden_size, 
                               handle->hidden_size);
        output_linear.push_back(linear);

        op_Batch_Matmul *batchgemm = new op_Batch_Matmul(handle);
        query_key.push_back(batchgemm);

        batchgemm = new op_Batch_Matmul(handle);
        head_value.push_back(batchgemm);

        op_BatchedLinear *batchlinear = new op_BatchedLinear(
                num_layer + "attention_self_query_kernel",
                num_layer + "attention_self_query_bias",
                num_layer + "attention_self_key_kernel",
                num_layer + "attention_self_key_bias",
                num_layer + "attention_self_value_kernel",
                num_layer + "attention_self_value_bias",
                handle, 
                handle->hidden_size * handle->hidden_size, 
                handle->hidden_size);
        batched_linear.push_back(batchlinear);

        op_FusionTranspose *trans = new op_FusionTranspose(handle);
        split_heads.push_back(trans);

        trans = new op_FusionTranspose(handle);
        merge_heads.push_back(trans);

        op_Mask_Add *op_mask = new op_Mask_Add(handle);
        mask.push_back(op_mask);

        op_Gelu *op_gelu = new op_Gelu(handle);
        gelu.push_back(op_gelu);

        if (handle->is_train) {
            if (handle->hidden_dropout_prob > 0 && handle->hidden_dropout_prob <= 1) {
                // BertOutput dropout
                op_Dropout *dropout = new op_Dropout(handle->hidden_dropout_prob, 
                                                     handle,
                                                     handle->max_train_batchsize * handle->max_train_seq_length * handle->hidden_size);
                output_dropout.push_back(dropout);

                // BertSelfOutput dropout
                dropout = new op_Dropout(handle->hidden_dropout_prob, 
                                         handle,
                                         handle->max_train_batchsize * handle->max_train_seq_length * handle->hidden_size);
                self_output_dropout.push_back(dropout);
            }

            if (handle->attention_probs_dropout_prob > 0 && handle->attention_probs_dropout_prob <= 1) {
                // BertSelfAttention dropout
                op_Dropout *dropout = new op_Dropout(handle->attention_probs_dropout_prob,
                                                     handle,
                                                     handle->max_train_batchsize * handle->num_attention_heads
                                                     * handle->max_train_seq_length * handle->max_train_seq_length);
                self_attention_dropout.push_back(dropout);
            }
        }
    }


    pooler_linear = new op_Linear("pooler_dense_kernel",
                                  "pooler_dense_bias",
                                  handle, 
                                  handle->hidden_size * handle->hidden_size, 
                                  handle->hidden_size);

    classify_linear = new op_Linear("classifier_kernel",
                                    "classifier_bias",
                                    handle, 
                                    handle->hidden_size * handle->num_classes, 
                                    handle->num_classes);
    
    loss = new op_CrossEntropyLoss(handle);

    classify_softmax = new op_SoftMax(handle);

    embedding = new Embedding(handle);

    if (handle->is_train && handle->hidden_dropout_prob > 0 && handle->hidden_dropout_prob <= 1) {
        // pooler dropout
        pooler_dropout = new op_Dropout(handle->hidden_dropout_prob, 
                                        handle,
                                        handle->max_train_batchsize * handle->hidden_size);
        // embedding dropout
        embedding_dropout = new op_Dropout(handle->hidden_dropout_prob, 
                                           handle,
                                           handle->max_train_batchsize * handle->max_train_seq_length * handle->hidden_size);
    }

    op_tanh = new op_Tanh(handle);

}

void bert::update_lr_start(double lr) {
    handle->update_learning_rate = true;
    handle->learning_rate = lr;
}

void bert::update_lr_end() {
    handle->update_learning_rate = false;
}

void bert::copy_inputs(int *&words,
                       int *&token_type,
                       int *&position,
                       int *&attention_mask) {
    size_t batchsize = handle->batchsize;
    size_t seq_length = handle->seq_length;

    int total_length = batchsize * seq_length;
    int *word_gpu, *token_type_gpu, *positions_gpu, *mask_gpu;

    int positions[total_length];
    for (int i = 0; i < total_length; i++) {
        positions[i] = i % seq_length;
    }

    int *host_input_package;
    checkCudaErrors(hipHostMalloc((void **) &host_input_package, 4 * total_length * sizeof(int)));
    memcpy(host_input_package, words, total_length * sizeof(int));
    memcpy(host_input_package + total_length, token_type, total_length * sizeof(int));
    memcpy(host_input_package + 2 * total_length, positions, total_length * sizeof(int));

    word_gpu = handle->global_malloc_manage_int.get_new_head_point(total_length);
    token_type_gpu = handle->global_malloc_manage_int.get_new_head_point(total_length);
    positions_gpu = handle->global_malloc_manage_int.get_new_head_point(total_length);
    if (attention_mask != nullptr) {
        mask_gpu = handle->global_malloc_manage_int.get_new_head_point(total_length);
        memcpy(host_input_package + 3 * total_length, attention_mask, total_length * sizeof(int));
        checkCudaErrors(hipMemcpyAsync(word_gpu,
                                        host_input_package,
                                        4 * total_length * sizeof(int),
                                        hipMemcpyHostToDevice));
        attention_mask = mask_gpu;
    } else {
        checkCudaErrors(hipMemcpyAsync(word_gpu,
                                        host_input_package,
                                        3 * total_length * sizeof(int),
                                        hipMemcpyHostToDevice));
    }
    hipHostFree(host_input_package);
    words = word_gpu;
    token_type = token_type_gpu;
    position = positions_gpu;
}

void bert::BERT_Inference(
        int *words,
        int *token_types,
        size_t batchsize,
        size_t seq_length,
        int *attention_mask) {

    size_t hidden_size = handle->hidden_size;
    size_t total_length = batchsize * seq_length * hidden_size;
    size_t num_words = batchsize * seq_length;
    size_t num_attention_heads = handle->num_attention_heads;
    size_t intermediate_size = handle->intermediate_size;

    handle->set_scale(batchsize, seq_length);
    handle->reset();

    int *positions;
    copy_inputs(words, token_types, positions, attention_mask);

    float *embedding_out;

    embedding->forward(embedding_out, words, token_types, positions);

    // Embedding output
//    debug_tensor_gpu<float>(std::string("embedding_out"), embedding_out, 3, handle->hidden_size, handle->batchsize * handle->seq_length);

    float *tensor_layer = embedding_out, *temp;

    for (int i = 0; i < handle->num_hidden_layers; i++) {
        handle->global_malloc_manage_float.record_layer_start();

        // start of Attention

        float *batched_gemm_out, *split_heads_out;
        batched_linear[i]->forward(batched_gemm_out,
                                   tensor_layer,
                                   batchsize * seq_length,
                                   hidden_size,
                                   hidden_size);

        split_heads[i]->forward(split_heads_out, batched_gemm_out, 3, true);

        float *head_query, *head_key, *head_val;
        head_query = split_heads_out;
        head_key = head_query + total_length;
        head_val = head_key + total_length;


        float *query_key_gemm;
        query_key[i]->forward(batchsize * num_attention_heads,
                              seq_length,
                              hidden_size / num_attention_heads,
                              seq_length,
                              head_query,
                              head_key,
                              query_key_gemm,
                              false,
                              true);

        mask[i]->forward(query_key_gemm, attention_mask, sqrt(handle->hidden_size / handle->num_attention_heads));

        softmax[i]->forward(query_key_gemm,
                            batchsize * num_attention_heads * seq_length,
                            seq_length);

        float *attention;
        head_value[i]->forward(batchsize * num_attention_heads,
                               seq_length,
                               seq_length,
                               hidden_size / num_attention_heads,
                               query_key_gemm,
                               head_val,
                               attention,
                               false,
                               false);

        float *merge_heads_out;
        merge_heads[i]->forward(merge_heads_out, attention, 1, false);

        attention_linear[i]->forward(temp,
                                     merge_heads_out,
                                     num_words,
                                     hidden_size,
                                     hidden_size);
        merge_heads_out = temp;

        attention_layernorm[i]->forward(tensor_layer,
                                        tensor_layer,
                                        num_words,
                                        hidden_size,
                                        merge_heads_out);

        // End of Attention
        // Start of Intermediate

        float *intermediate_out;
        intermediate_linear[i]->forward(intermediate_out,
                                        tensor_layer,
                                        num_words,
                                        hidden_size,
                                        intermediate_size);

        gelu[i]->forward(intermediate_out, num_words * intermediate_size);

        // End of Intermedaite
        // Start of Output

        float *output_out;
        output_linear[i]->forward(output_out,
                                  intermediate_out,
                                  num_words,
                                  intermediate_size,
                                  hidden_size);

        output_layernorm[i]->forward(tensor_layer,
                                     tensor_layer,
                                     num_words,
                                     hidden_size,
                                     output_out);

        hipEventRecord(handle->layer_compute_done, handle->cal_stream);
        hipEventSynchronize(handle->layer_compute_done);
        handle->global_malloc_manage_float.reuse_layer_mem();
        //  Layer End
    }
    // Pooler Start
    float *first_token, *pooler_out;
    copy_pooler(first_token, tensor_layer, handle);

    pooler_linear->forward(pooler_out,
                           first_token,
                           batchsize,
                           hidden_size,
                           hidden_size);

//    debug_tensor_gpu<float>(std::string("pooler_out"), pooler_out, 10, handle->hidden_size, 1);

    op_tanh->forward(pooler_out, batchsize * hidden_size);
//    debug_tensor_gpu<float>(std::string("pooler_out"), pooler_out, 10, handle->hidden_size, 1);

    // Pooler End

    ret.tensor = tensor_layer;
    ret.pooled_output = pooler_out;
}

float *bert::classify_inference(float *pooler_out, size_t num_classes) {
    float *classify_out;
    classify_linear->forward(classify_out,
                             pooler_out,
                             handle->batchsize,
                             handle->hidden_size,
                             num_classes);

    classify_softmax->forward(classify_out, handle->batchsize, num_classes);

    return classify_out;
}

void bert::BERT_train_forward(
        int *words,
        int *token_types,
        size_t batchsize,
        size_t seq_length,
        int *attention_mask) {

    size_t hidden_size = handle->hidden_size;
    size_t total_length = batchsize * seq_length * hidden_size;
    size_t num_words = batchsize * seq_length;
    size_t num_attention_heads = handle->num_attention_heads;
    size_t intermediate_size = handle->intermediate_size;

    handle->set_scale(batchsize, seq_length);

    int *positions;
    copy_inputs(words, token_types, positions, attention_mask);

    float *embedding_out;

    embedding->forward(embedding_out, words, token_types, positions);
    // Embedding output

    float *tensor_layer = embedding_out, *temp;
    float *embedding_dropout_out, *self_attention_dropout_out, *self_output_dropout_out;

    if (handle->is_train && handle->hidden_dropout_prob > 0 && handle->hidden_dropout_prob <= 1) {
        embedding_dropout->forward(embedding_dropout_out,
                                   embedding_out,
                                   handle->batchsize * handle->seq_length * handle->hidden_size);
        tensor_layer = embedding_dropout_out;
    }
    for (int i = 0; i < handle->num_hidden_layers; i++) {
        // start of Attention
        float *batched_gemm_out, *split_heads_out;

        batched_linear[i]->forward(batched_gemm_out,
                                   tensor_layer,
                                   batchsize * seq_length,
                                   hidden_size,
                                   hidden_size);

        split_heads[i]->forward(split_heads_out, batched_gemm_out, 3, true);

        float *head_query, *head_key, *head_val;
        head_query = split_heads_out;
        head_key = head_query + total_length;
        head_val = head_key + total_length;

        float *query_key_gemm;
        query_key[i]->forward(batchsize * num_attention_heads,
                              seq_length,
                              hidden_size / num_attention_heads,
                              seq_length,
                              head_query,
                              head_key,
                              query_key_gemm,
                              false,
                              true);

        mask[i]->forward(query_key_gemm, 
                         attention_mask, 
                         sqrt(handle->hidden_size / handle->num_attention_heads));

        softmax[i]->forward(query_key_gemm,
                            batchsize * num_attention_heads * seq_length,
                            seq_length);

        // TODO: BertSelfAttention dropout
        if (handle->is_train && handle->attention_probs_dropout_prob > 0 && handle->attention_probs_dropout_prob <= 1) {
            self_attention_dropout[i]->forward(self_attention_dropout_out, 
                                               query_key_gemm,
                                               handle->batchsize * handle->num_attention_heads * 
                                                    handle->seq_length * handle->seq_length);
            query_key_gemm = self_attention_dropout_out;
        }

        float *attention;
        head_value[i]->forward(batchsize * num_attention_heads,
                               seq_length,
                               seq_length,
                               hidden_size / num_attention_heads,
                               query_key_gemm,
                               head_val,
                               attention,
                               false,
                               false);

        float *merge_heads_out;
        merge_heads[i]->forward(merge_heads_out, attention, 1, false);

        attention_linear[i]->forward(temp,
                                     merge_heads_out,
                                     num_words,
                                     hidden_size,
                                     hidden_size);
        merge_heads_out = temp;

        // TODO: BertSelfOutput dropout
        if (handle->is_train && handle->hidden_dropout_prob > 0 && handle->hidden_dropout_prob <= 1) {
            self_output_dropout[i]->forward(self_output_dropout_out, 
                                            merge_heads_out,
                                            handle->batchsize * handle->seq_length * handle->hidden_size);
            merge_heads_out = self_output_dropout_out;
        }

        float *attention_layernorm_out;
        attention_layernorm[i]->forward(attention_layernorm_out,
                                        merge_heads_out,
                                        num_words,
                                        hidden_size,
                                        tensor_layer);

        // End of Attention
        // Start of Intermediate
        float *intermediate_out;
        intermediate_linear[i]->forward(intermediate_out,
                                        attention_layernorm_out,
                                        num_words,
                                        hidden_size,
                                        intermediate_size);

        gelu[i]->forward(intermediate_out, num_words * intermediate_size);

        // End of Intermedaite
        // Start of Output
        float *output_out;
        output_linear[i]->forward(output_out,
                                  intermediate_out,
                                  num_words,
                                  intermediate_size,
                                  hidden_size);

        // TODO: output_dropout
        if (handle->is_train && handle->hidden_dropout_prob > 0 && handle->hidden_dropout_prob <= 1) {
            float *output_dropout_out;
            output_dropout[i]->forward(output_dropout_out,
                                       output_out,
                                       handle->batchsize * handle->seq_length * handle->hidden_size);
            output_out = output_dropout_out;
        }


        float *output_layernorm_out;
        output_layernorm[i]->forward(output_layernorm_out,
                                     output_out,
                                     num_words,
                                     hidden_size,
                                     attention_layernorm_out);

//        debug_tensor_gpu<float>(std::string("output_layernorm[i]->gamma"), output_layernorm[i]->gamma, 1, 1, 1);

        tensor_layer = output_layernorm_out;
        //  Layer End
    }
    // Pooler Start
    float *first_token, *pooler_out;
    copy_pooler(first_token, tensor_layer, handle);

    pooler_linear->forward(pooler_out,
                           first_token,
                           batchsize,
                           hidden_size,
                           hidden_size);

    op_tanh->forward(pooler_out, batchsize * hidden_size);

    // Pooler End

    ret.tensor = tensor_layer;
    ret.pooled_output = pooler_out;
}

float bert::classify_train(int *classes, float *pooler_out, size_t num_classes) {
    float *loss_out;
    float *classify_out;

    if (handle->is_train && handle->hidden_dropout_prob > 0 && handle->hidden_dropout_prob <= 1) {
        float *pooler_dropout_out;
        pooler_dropout->forward(pooler_dropout_out, 
                                pooler_out,
                                handle->batchsize * handle->hidden_size);
        pooler_out = pooler_dropout_out;
    }

    classify_linear->forward(classify_out,
                             pooler_out,
                             handle->batchsize,
                             handle->hidden_size,
                             num_classes);
//    debug_tensor_gpu<float>(std::string("classify_out"), classify_out, 2, 2, handle->batchsize);

    int *calsses_gpu;
    calsses_gpu = handle->global_malloc_manage_int.get_new_head_point(handle->batchsize);
    checkCudaErrors(hipMemcpyAsync(calsses_gpu, classes, handle->batchsize * sizeof(int), hipMemcpyHostToDevice));

    loss->forward(loss_out, classify_out, calsses_gpu, handle->batchsize, num_classes);
//    debug_tensor_gpu<float>(std::string("loss_out"), loss_out, handle->batchsize + 1, handle->batchsize + 1);

    float *cpu_mem;
    cpu_mem = (float *) malloc(sizeof(float) * 1);
    checkCudaErrors(hipMemcpyAsync(cpu_mem, loss_out + handle->batchsize, sizeof(float) * 1, hipMemcpyDeviceToHost));

    float loss_return = (*cpu_mem);

    if (!handle->is_train)
        return loss_return;
    float *dout_gpu;
    dout_gpu = handle->global_malloc_manage_float.get_new_head_point(1);
    float *dout = (float *) malloc(sizeof(float));
    dout[0] = 1.0;
    checkCudaErrors(hipMemcpyAsync(dout_gpu, dout, sizeof(float), hipMemcpyHostToDevice));

    loss->backward(dout_gpu, handle->batchsize, num_classes, calsses_gpu);

    classify_linear->backward(loss->grad_input, 
                              handle->batchsize,
                              handle->hidden_size,
                              num_classes);

    float *deal_dropout = classify_linear->grad_input;
    if (handle->is_train && handle->hidden_dropout_prob > 0 && handle->hidden_dropout_prob <= 1) {
        pooler_dropout->backward(classify_linear->grad_input);
        deal_dropout = pooler_dropout->grad_input;
    }

    op_tanh->backward(deal_dropout, handle->batchsize * handle->hidden_size);
    pooler_linear->backward(op_tanh->grad_input, handle->batchsize, handle->hidden_size, handle->hidden_size);

    float *copy_pooler_grad_input;
    copy_pooler_backward(copy_pooler_grad_input, pooler_linear->grad_input, handle);

    float *tensor_layer_grad_input = copy_pooler_grad_input;

    for (int i = handle->num_hidden_layers - 1; i >= 0; i--) {
        if (handle->optim_running_time)
            handle->global_malloc_manage_float.record_layer_start();

        output_layernorm[i]->backward(tensor_layer_grad_input, 
                                      handle->batchsize * handle->seq_length,
                                      handle->hidden_size);

        deal_dropout = output_layernorm[i]->grad_input;
        if (handle->is_train && handle->hidden_dropout_prob > 0 && handle->hidden_dropout_prob <= 1) {
            output_dropout[i]->backward(output_layernorm[i]->grad_input);
            deal_dropout = output_dropout[i]->grad_input;
        }

        output_linear[i]->backward(deal_dropout, 
                                   handle->batchsize * handle->seq_length,
                                   handle->intermediate_size, 
                                   handle->hidden_size);

        gelu[i]->backward(output_linear[i]->grad_input,
                          handle->batchsize * handle->seq_length * handle->intermediate_size);

        intermediate_linear[i]->backward(gelu[i]->grad_input,
                                         handle->batchsize * handle->seq_length,
                                         handle->hidden_size, 
                                         handle->intermediate_size);

        short_cut_backward(intermediate_linear[i]->grad_input, 
                           output_layernorm[i]->grad_input,
                           handle->batchsize * handle->seq_length * handle->hidden_size,
                           handle);

        attention_layernorm[i]->backward(intermediate_linear[i]->grad_input, 
                                         handle->batchsize * handle->seq_length,
                                         handle->hidden_size);

        deal_dropout = attention_layernorm[i]->grad_input;
        if (handle->is_train && handle->hidden_dropout_prob > 0 && handle->hidden_dropout_prob <= 1) {
            self_output_dropout[i]->backward(attention_layernorm[i]->grad_input);
            deal_dropout = self_output_dropout[i]->grad_input;
        }

        attention_linear[i]->backward(deal_dropout, 
                                      handle->batchsize * handle->seq_length,
                                      handle->hidden_size, 
                                      handle->hidden_size);

        merge_heads[i]->backward(attention_linear[i]->grad_input, 1, false);

        head_value[i]->backward(merge_heads[i]->grad_input,
                                handle->batchsize * handle->num_attention_heads,
                                handle->seq_length, 
                                handle->seq_length,
                                handle->hidden_size / handle->num_attention_heads);

        deal_dropout = head_value[i]->grad_input;
        if (handle->is_train && handle->attention_probs_dropout_prob > 0 && handle->attention_probs_dropout_prob <= 1) {
            self_attention_dropout[i]->backward(head_value[i]->grad_input);
            deal_dropout = self_attention_dropout[i]->grad_input;
        }


        softmax[i]->backward(deal_dropout,
                             handle->batchsize * handle->num_attention_heads * handle->seq_length,
                             handle->seq_length);

        mask[i]->backward(softmax[i]->grad_input,
                          handle->seq_length * handle->seq_length * handle->batchsize * handle->num_attention_heads,
                          (float) 1.0 / sqrt(handle->hidden_size / handle->num_attention_heads));

        query_key[i]->backward(mask[i]->grad_query_key_gemm, handle->batchsize * handle->num_attention_heads,
                               handle->seq_length,
                               handle->hidden_size / handle->num_attention_heads,
                               handle->seq_length, false, true);

        size_t tot_length = handle->batchsize * handle->seq_length * handle->hidden_size;
        float *dout1 = handle->global_malloc_manage_float.get_new_head_point(
                tot_length * 3);

//      query, key, value
        {
            dim3 threads(1024, 1, 1);
            dim3 blocks(min((long) 65535, tot_length * 3 / 1024) + 1, 1, 1);
            MemoryCpyLinear<float> << < blocks, threads, 0, handle->cal_stream >> > (
                    dout1, query_key[i]->grad_input, tot_length, tot_length);
            MemoryCpyLinear<float> << < blocks, threads, 0, handle->cal_stream >> > (
                    dout1 + tot_length, query_key[i]->grad_kernel, tot_length, tot_length);
            MemoryCpyLinear<float> << < blocks, threads, 0, handle->cal_stream >> > (
                    dout1 + 2 * tot_length, head_value[i]->grad_kernel, tot_length, tot_length);
        }

        merge_heads[i]->backward(dout1, 3, true);

        batched_linear[i]->backward(merge_heads[i]->grad_input, 
                                    attention_layernorm[i]->grad_input,
                                    handle->batchsize * handle->seq_length, 
                                    handle->hidden_size, 
                                    handle->hidden_size);

        tensor_layer_grad_input = batched_linear[i]->grad_input;
        
        if (handle->optim_running_time)
            handle->global_malloc_manage_float.reuse_layer_mem();
    }

    if (handle->is_train && handle->hidden_dropout_prob > 0 && handle->hidden_dropout_prob <= 1) {
        embedding_dropout->backward(tensor_layer_grad_input);
        tensor_layer_grad_input = embedding_dropout->grad_input;
    }

    embedding->backward(tensor_layer_grad_input);

    return loss_return;
}