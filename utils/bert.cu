#include "bert.cuh"
#include "load_model.h"

bert::bert (bool BERT_Large, int num_gpu, std::string dir) {
    checkCudaErrors(hipSetDevice(num_gpu));
    handle = new global_handle(BERT_Large, dir);
    init_ops();
}

void bert::init_ops(){

    for(int i = 0; i < handle->num_hidden_layers; i++){
        std::string num_layer = "_" + std::to_string(i) + "_";

        op_LayerNorm* layernorm = new op_LayerNorm(num_layer + "attention_output_LayerNorm_gamma",
                                                num_layer + "attention_output_LayerNorm_beta",
                                                handle);
        attention_layernorm.push_back(layernorm);

        layernorm = new op_LayerNorm(num_layer + "output_LayerNorm_gamma",
                                    num_layer + "output_LayerNorm_beta",
                                    handle);
        output_layernorm.push_back(layernorm);

        op_SoftMax* Softmax = new op_SoftMax(handle);
        softmax.push_back(Softmax);

        op_Linear* linear = new op_Linear(num_layer + "attention_output_dense_kernel",
                                          num_layer + "attention_output_dense_bias",
                                           handle);
        attention_linear.push_back(linear);
        
        linear = new op_Linear(num_layer + "intermediate_dense_kernel",
                                num_layer + "intermediate_dense_bias",
                                handle);
        intermediate_linear.push_back(linear);

        linear = new op_Linear(num_layer + "output_dense_kernel",
                                num_layer + "output_dense_bias",
                                handle);
        output_linear.push_back(linear);

        op_Batch_Matmul* batchgemm = new op_Batch_Matmul(handle);
        query_key.push_back(batchgemm);

        batchgemm = new op_Batch_Matmul(handle);
        head_value.push_back(batchgemm);

        op_BatchedLinear* batchlinear = new op_BatchedLinear(
                                        num_layer + "attention_self_query_kernel",
                                        num_layer + "attention_self_query_bias",
                                        num_layer + "attention_self_key_kernel",
                                        num_layer + "attention_self_key_bias",
                                        num_layer + "attention_self_value_kernel",
                                        num_layer + "attention_self_value_bias",
                                        handle);
        batched_linear.push_back(batchlinear);

        op_FusionTranspose* trans = new op_FusionTranspose(handle);
        split_heads.push_back(trans);

        trans = new op_FusionTranspose(handle);
        merge_heads.push_back(trans);

        op_Mask_Add* op_mask = new op_Mask_Add(handle);
        mask.push_back(op_mask);

        op_Gelu* op_gelu = new op_Gelu(handle);
        gelu.push_back(op_gelu);
    }

    pooler_linear = new op_Linear(  "pooler_dense_kernel",
                                    "pooler_dense_bias",
                                    handle);
    
    classify_linear = new op_Linear("classifier_kernel",
                                    "classifier_bias",
                                    handle);
    
    classify_softmax = new op_SoftMax(handle);

    embedding = new Embedding(handle);

    op_tanh = new op_Tanh(handle);
}

void bert::copy_inputs( int* &words, 
                        int* &token_type,
                        int* &position,
                        int* &attention_mask){
    size_t batchsize = handle->batchsize;
    size_t seq_length = handle->seq_length;

    int total_length = batchsize * seq_length;
    int *word_gpu, *token_type_gpu, *positions_gpu, *mask_gpu;

    int positions[total_length];
    for( int i = 0; i < total_length; i++){
        positions[i] = i % seq_length;
    }
    
    int* host_input_package;
    checkCudaErrors(hipHostMalloc((void **)&host_input_package, 4*total_length*sizeof(int)));
    memcpy(host_input_package, words, total_length*sizeof(int));
    memcpy(host_input_package + total_length, token_type, total_length*sizeof(int));
    memcpy(host_input_package + 2*total_length, positions, total_length*sizeof(int));

    word_gpu = handle->global_malloc_manage_int.get_new_head_point(total_length);
    token_type_gpu = handle->global_malloc_manage_int.get_new_head_point(total_length);
    positions_gpu = handle->global_malloc_manage_int.get_new_head_point(total_length);
    if(attention_mask != nullptr){
        mask_gpu = handle->global_malloc_manage_int.get_new_head_point(total_length);
        memcpy(host_input_package + 3*total_length, attention_mask, total_length*sizeof(int));
        checkCudaErrors(hipMemcpyAsync(word_gpu, 
                                        host_input_package, 
                                        4*total_length*sizeof(int), 
                                        hipMemcpyHostToDevice));
        attention_mask = mask_gpu;
    }
    else{
        checkCudaErrors(hipMemcpyAsync(word_gpu, 
                                        host_input_package, 
                                        3*total_length*sizeof(int), 
                                        hipMemcpyHostToDevice));
    }
    hipHostFree(host_input_package);
    words = word_gpu;
    token_type = token_type_gpu;
    position = positions_gpu;
}

void bert::BERT_Inference (
                    int* words, 
                    int* token_types, 
                    size_t batchsize, 
                    size_t seq_length, 
                    int* attention_mask){

    size_t hidden_size = handle->hidden_size;
    size_t total_length = batchsize * seq_length * hidden_size;
    size_t num_words = batchsize * seq_length;
    size_t num_attention_heads= handle->num_attention_heads;
    size_t intermediate_size = handle->intermediate_size;

    handle->set_scale(batchsize, seq_length);
    handle->reset();

    int* positions;
    copy_inputs(words, token_types, positions, attention_mask);

    float *embedding_out;

    embedding->forward(embedding_out, words, token_types, positions);

    float *tensor_layer = embedding_out, *temp;

    for(int i = 0; i < handle->num_hidden_layers; i++){

        handle->global_malloc_manage_float.record_layer_start();
        
        // start of Attention

        float *batched_gemm_out, *split_heads_out;
        batched_linear[i]->forward(batched_gemm_out,
                                   tensor_layer,
                                   batchsize * seq_length,
                                   hidden_size,
                                   hidden_size);
        
        split_heads[i]->forward(split_heads_out, batched_gemm_out, 3, true);

        float *head_query, *head_key, *head_val;
        head_query = split_heads_out;
        head_key = head_query + total_length;
        head_val = head_key + total_length;


        float *query_key_gemm;
        query_key[i]->forward(batchsize * num_attention_heads,
                            seq_length,
                            hidden_size / num_attention_heads,
                            seq_length,
                            head_query,
                            head_key,
                            query_key_gemm,
                            false,
                            true);
        
        mask[i]->forward(query_key_gemm, attention_mask, 8.0);
        
        softmax[i]->forward(query_key_gemm,
                            batchsize * num_attention_heads * seq_length,
                            seq_length);
        
        float* attention;
        head_value[i]->forward(batchsize * num_attention_heads,
                             seq_length,
                             seq_length,
                             hidden_size / num_attention_heads,
                             query_key_gemm,
                             head_val,
                             attention,
                             false,
                             false);
        
        float *merge_heads_out;
        merge_heads[i]->forward(merge_heads_out, attention, 1, false);
        
        attention_linear[i]->forward(temp,
                                     merge_heads_out,
                                     num_words,
                                     hidden_size,
                                     hidden_size);
        merge_heads_out = temp;
        
        attention_layernorm[i]->forward(tensor_layer,
                                        tensor_layer,
                                        num_words,
                                        hidden_size,
                                        merge_heads_out);
                
        // End of Attention
        // Start of Intermediate

        float* intermediate_out;
        intermediate_linear[i]->forward(intermediate_out,
                                        tensor_layer,
                                        num_words,
                                        hidden_size,
                                        intermediate_size);

        gelu[i]->forward(intermediate_out, num_words * intermediate_size);

        // End of Intermedaite
        // Start of Output

        float* output_out;
        output_linear[i]->forward(output_out,
                                  intermediate_out,
                                  num_words,
                                  intermediate_size,
                                  hidden_size);

        output_layernorm[i]->forward(tensor_layer,
                                     tensor_layer,
                                     num_words,
                                     hidden_size,
                                     output_out);

        hipStreamSynchronize(handle->cal_stream);
        handle->global_malloc_manage_float.reuse_layer_mem();
        //  Layer End
    }
    // Pooler Start
    float* first_token, *pooler_out;
    copy_pooler(first_token, tensor_layer, handle);

    pooler_linear->forward(pooler_out,
                           first_token,
                           batchsize,
                           hidden_size,
                           hidden_size);
    
    op_tanh->forward(pooler_out, batchsize * hidden_size);
    // Pooler End
    
    ret.tensor = tensor_layer;
    ret.pooled_output = pooler_out;
}

float* bert::classify_inference(float* pooler_out, size_t num_classes){
    float* classify_out;
    classify_linear->forward(classify_out,
                             pooler_out,
                             handle->batchsize,
                             handle->hidden_size,
                             num_classes);
    
    classify_softmax->forward(classify_out, handle->batchsize, num_classes);

    return classify_out;
}
