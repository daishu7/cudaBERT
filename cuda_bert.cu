#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <time.h>
#include <iostream>
#include <iomanip>

#include "utils/common.h"
#include "utils/bert.cuh"
#include "utils/tokenization.h"
#include "cuda_bert.cuh"

int *filling_inputs(int *tensor, int seq_length, int start_length, int batchsize) {
    int *target = (int *) malloc(sizeof(int) * seq_length * batchsize);
    for (int i = 0; i < seq_length - 1; i++) {
        target[i] = tensor[i % (start_length - 1)];
    }
    target[seq_length - 1] = tensor[start_length - 1];
    for (int i = 1; i < batchsize; i++) {
        memcpy(target + seq_length * i, target, seq_length * sizeof(int));
    }
    return target;
}

int *anthor_filling_inputs(int *tensor, int seq_length, int start_length, int batchsize) {
    int *target = (int *) malloc(sizeof(int) * seq_length * batchsize);
    for (int i = 0; i < start_length; i++) {
        target[i] = tensor[i];
    }
    for (int i = start_length; i < seq_length; i++) {
        target[i] = 0;
    }
    for (int i = 1; i < batchsize; i++) {
        memcpy(target + seq_length * i, target, seq_length * sizeof(int));
    }
    return target;
}

void *cubert_open_tokenizer(const char *vocab_file, bool do_lower_case) {
    return new FullTokenizer(vocab_file, do_lower_case);
}

void cubert_close_tokenizer(void *tokenizer) {
    delete (FullTokenizer *) tokenizer;
}

/**
 * Truncates a sequence pair in place to the maximum length.
 * @param tokens_a
 * @param tokens_a
 * @param max_length
 */
void _truncate_seq_pair(std::vector <std::string> *tokens_a,
                        std::vector <std::string> *tokens_b,
                        size_t max_length) {
// This is a simple heuristic which will always truncate the longer sequence
// one token at a time. This makes more sense than truncating an equal percent
// of tokens from each, since if one sequence is very short then each token
// that's truncated likely contains more information than a longer sequence.
    while (true) {
        size_t total_length = tokens_a->size() + tokens_b->size();
        if (total_length <= max_length) {
            break;
        }
        if (tokens_a->size() > tokens_b->size()) {
            tokens_a->pop_back();
        } else {
            tokens_b->pop_back();
        }
    }
}

/**
 * Converts a single `InputExample` into a single `InputFeatures`.
 */
void convert_single_example(FullTokenizer *tokenizer,
                            size_t max_seq_length,
                            const char *text_a, const char *text_b,
                            int *input_ids, int *input_mask, int *segment_ids) {
    std::vector <std::string> tokens_a;
    tokens_a.reserve(max_seq_length);

    std::vector <std::string> tokens_b;
    tokens_b.reserve(max_seq_length);

    tokenizer->tokenize(text_a, &tokens_a, max_seq_length);
    if (text_b != nullptr) {
        tokenizer->tokenize(text_b, &tokens_b, max_seq_length);

        // Modifies `tokens_a` and `tokens_b` in place so that the total
        // length is less than the specified length.
        // Account for [CLS], [SEP], [SEP] with "- 3"
        _truncate_seq_pair(&tokens_a, &tokens_b, max_seq_length - 3);
    } else {
        if (tokens_a.size() > max_seq_length - 2) {
            tokens_a.resize(max_seq_length - 2);
        }
    }

    // The convention in BERT is:
    // (a) For sequence pairs:
    //  tokens:   [CLS] is this jack ##son ##ville ? [SEP] no it is not . [SEP]
    //  type_ids: 0     0  0    0    0     0       0 0     1  1  1  1   1 1
    // (b) For single sequences:
    //  tokens:   [CLS] the dog is hairy . [SEP]
    //  type_ids: 0     0   0   0  0     0 0
    //
    // Where "type_ids" are used to indicate whether this is the first
    // sequence or the second sequence. The embedding vectors for `type=0` and
    // `type=1` were learned during pre-training and are added to the wordpiece
    // embedding vector (and position vector). This is not *strictly* necessary
    // since the [SEP] token unambiguously separates the sequences, but it makes
    // it easier for the model to learn the concept of sequences.
    //
    // For classification tasks, the first vector (corresponding to [CLS]) is
    // used as as the "sentence vector". Note that this only makes sense because
    // the entire model is fine-tuned.
    input_ids[0] = tokenizer->convert_token_to_id("[CLS]");
    segment_ids[0] = 0;
    for (int i = 0; i < tokens_a.size(); ++i) {
        input_ids[i + 1] = tokenizer->convert_token_to_id(tokens_a[i]);
        segment_ids[i + 1] = 0;
    }
    input_ids[tokens_a.size() + 1] = tokenizer->convert_token_to_id("[SEP]");
    segment_ids[tokens_a.size() + 1] = 0;

    if (text_b != nullptr) {
        for (int i = 0; i < tokens_b.size(); ++i) {
            input_ids[i + tokens_a.size() + 2] = tokenizer->convert_token_to_id(tokens_b[i]);
            segment_ids[i + tokens_a.size() + 2] = 1;
        }
        input_ids[tokens_b.size() + tokens_a.size() + 2] = tokenizer->convert_token_to_id("[SEP]");
        segment_ids[tokens_b.size() + tokens_a.size() + 2] = 1;
    }

    size_t len = text_b != nullptr ? tokens_a.size() + tokens_b.size() + 3 : tokens_a.size() + 2;
    std::fill_n(input_mask, len, 1);

    // Zero-pad up to the sequence length.
    std::fill_n(input_ids + len, max_seq_length - len, 0);
    std::fill_n(input_mask + len, max_seq_length - len, 0);
    std::fill_n(segment_ids + len, max_seq_length - len, 0);
}

void convert_batch_example(void *tokenizer, int batch_size,
                           int max_seq_length,
                           std::vector<std::string> &text_a,
                           std::vector<std::string> &text_b,
                           std::vector<int> &gt_classes,
                           std::vector<int> &text_id,
                           int *input_ids, int *segment_ids, int *input_mask, int *classes) {
    //TODO:Cut string len

    for (int batch_idx = 0; batch_idx < text_id.size(); ++batch_idx) {
        classes[batch_idx] = gt_classes[text_id[batch_idx]];
        convert_single_example((FullTokenizer *) tokenizer,
                               max_seq_length,
                               text_a[text_id[batch_idx]].c_str(),
                               text_b.size() == 0 ? nullptr : text_b[text_id[batch_idx]].c_str(),
                               input_ids + max_seq_length * batch_idx,
                               input_mask + max_seq_length * batch_idx,
                               segment_ids + max_seq_length * batch_idx);
    }
}


extern "C" {

bert *init_model(bool large = false, int num_gpu = 0, std::string dir = "", bool is_train = false, float lr = 0.001,
                 std::string optim = "sgd", bool optimRunningTime = true, int num_classes = 2) {
    bert *ret = new bert(large, num_gpu, dir, is_train, optimRunningTime, num_classes, optim, lr);
    return ret;
}

Retval Cuda_Inference(bert *model,
                      int *words,
                      int *token_types,
                      int batchsize,
                      int seq_length,
                      int *masks) {
    model->BERT_Inference(words,
                          token_types,
                          batchsize,
                          seq_length,
                          masks);

    return model->ret;
}

void Cuda_Classify(bert *model,
                   float *output,
                   int *words,
                   int *token_types,
                   int batchsize,
                   int seq_length,
                   int num_classes,
                   int *attention_mask) {
    model->BERT_Inference(words,
                          token_types,
                          batchsize,
                          seq_length,
                          attention_mask);
    float *output_gpu;
    output_gpu = model->classify_inference(model->ret.pooled_output, num_classes);
    model->get_gpu_result(output, output_gpu, batchsize * num_classes);
}

float cuda_classify_train(bert *model,
                         int *words,
                         int *token_types,
                         int *classes,
                         int batchsize,
                         int seq_length,
                         int num_classes,
                         int *attention_mask) {
    model->BERT_train(words,
                      token_types,
                      batchsize,
                      seq_length,
                      attention_mask);
    float output_gpu = model->classify_train(classes, model->ret.pooled_output, num_classes);
    return output_gpu;
}

void test_inference(int batchsize, int seq_length, int nIter, bool base, int num_gpu) {
    bert *model = init_model(base, num_gpu);

    int test_word_id_seed[11] = {2040, 2001, 3958, 27227, 1029, 3958, 103,
                                 2001, 1037, 13997, 11510};
    int test_token_type_id_seed[11] = {0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1};

    int attention_mask[11] = {1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0};
    int classes[4] = {1, 1, 1, 1};

    int *test_word_id, *test_token_type_id, *test_attention_mask;
    test_word_id = anthor_filling_inputs(test_word_id_seed, seq_length, 11, batchsize);
    test_token_type_id = anthor_filling_inputs(test_token_type_id_seed, seq_length, 11, batchsize);
    test_attention_mask = anthor_filling_inputs(attention_mask, seq_length, 11, batchsize);
    std::cout << " Seq_length : " << seq_length << std::endl;
    std::cout << " Batchsize : " << batchsize << std::endl;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *output_pinned;
    checkCudaErrors(hipHostMalloc((void **) &output_pinned,
                                   (1024) * model->handle->hidden_size * sizeof(float)));

    //Warm Up
    for (int i = 0; i < 10; i++) {
        model->BERT_Inference(
                test_word_id,
                test_token_type_id,
                batchsize,
                seq_length,
                test_attention_mask);
        model->get_gpu_result(output_pinned,
                              model->ret.pooled_output,
                              model->handle->batchsize * model->handle->hidden_size);

        if (i == 0) {
            debug_tensor<float>(std::string("unit_test"),
                                output_pinned,
                                10,
                                model->handle->hidden_size,
                                max(model->handle->batchsize / 10, (long) 1));
        }
    }

    double total_time = 0;
    for (int i = 0; i < nIter; i++) {
        float it_time;
        hipEventRecord(start);
        float *output;
        // cuda_classify(
        //         model,
        //         output,
        //         test_word_id,
        //         test_token_type_id,
        //         classes,
        //         batchsize,
        //         seq_length,
        //         2,
        //         test_attention_mask
        // );
        model->BERT_Inference(
                test_word_id,
                test_token_type_id,
                batchsize,
                seq_length,
                test_attention_mask);

        model->get_gpu_result(output_pinned,
                              model->ret.pooled_output,
                              model->handle->batchsize * model->handle->hidden_size);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&it_time, start, stop);
        total_time += it_time;
    }

    delete model;

    double dSeconds = total_time / (double) nIter;
    printf("Time= %.2f(ms)\n", dSeconds);
}

int generateSeed(int i) {
    return rand() % i;
}

void bert_train(int batchsize, int seq_length, int nIter, bool base, int num_gpu) {

    bert *model = init_model(base, num_gpu, "", true);

    int input_ids[batchsize * seq_length];
    int input_mask[batchsize * seq_length];
    int segment_ids[batchsize * seq_length];
    int classes[batchsize];

    std::cout << " Seq_length : " << seq_length << std::endl;
    std::cout << " Batchsize : " << batchsize << std::endl;

    std::vector <std::string> text_a;
    std::vector <std::string> text_b;
    std::vector<int> gt_classes;
    read_tsv("/home/wenxh/zyc/bert_train/cuda_bert/data/deepqa_train_10w.tsv", text_a, gt_classes);

    int tot_line_len = text_a.size();
    srand(time(NULL));
    std::vector<int> text_id(tot_line_len);
    for (int i = 0; i < tot_line_len; i++)
        text_id[i] = i;
    void *tokenizer = cubert_open_tokenizer("/home/wenxh/zyc/bert_train/cuda_bert/data/vocab.txt", true);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double total_time = 0;
    double min_loss = 1e18;
    int num_labels = 2;
    int outputRand = 0;
    for (int i = 0; i < nIter; i++) {
//         TODO: random
//        random_shuffle(text_id.begin(), text_id.end(), generateSeed);
        double now_loss = 0;
        for (int j = 0; j < tot_line_len / batchsize + (tot_line_len % batchsize == 0 ? 0 : 1); j++) {
            std::vector<int> tmp_text_id;
            for (int k = j * batchsize; k < min((j + 1) * batchsize, tot_line_len); k++)
                tmp_text_id.push_back(text_id[k]);

            convert_batch_example(tokenizer, batchsize, seq_length, text_a, text_b, gt_classes, tmp_text_id, input_ids,
                                  segment_ids,
                                  input_mask, classes);

//            std::cout << "input_ids: " << std::endl;
//            for (int i = 0; i < batchsize * seq_length; i++) {
//                std::cout << input_ids[i] << " ";
//                if(i % seq_length == seq_length - 1)
//                    std::cout<< std::endl;
//            }
//            std::cout << std::endl;
//
//            std::cout << "segment_ids: " << std::endl;
//            for (int i = 0; i < batchsize * seq_length; i++) {
//                std::cout << segment_ids[i] << " ";
//                if(i % seq_length == seq_length - 1)
//                    std::cout<< std::endl;
//            }
//            std::cout << std::endl;
//
//            std::cout << "input_mask: " << std::endl;
//            for (int i = 0; i < batchsize * seq_length; i++) {
//                std::cout << input_mask[i] << " ";
//                if(i % seq_length == seq_length - 1)
//                    std::cout<< std::endl;
//            }
//            std::cout << std::endl;
//
//            std::cout << "label: " << std::endl;
//            for (int i = 0; i < batchsize; i++)
//                std::cout << classes[i] << " ";
//            std::cout << std::endl;

            float it_time;
            hipEventRecord(start);
            int tmp_batchsize = min( (j+1) * batchsize, tot_line_len) - j * batchsize;
            model->handle->batchsize = tmp_batchsize;

            float loss = cuda_classify_train(
                    model,
                    input_ids,
                    segment_ids,
                    classes,
                    tmp_batchsize,
                    seq_length,
                    num_labels,
                    input_mask
            );

            now_loss += loss * tmp_batchsize;

//            std::cout << "temp batch loss: " << (*cpu_mem) << std::endl;
            if(j % 200 == 199) {
                std::cout << "outputRand: " << outputRand << std::endl;
                std::cout << "average loss: " << std::fixed << std::setprecision(10) << now_loss / min( (j+1) * batchsize, tot_line_len) << std::endl;
                outputRand++;
            }
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&it_time, start, stop);
            total_time += it_time;
        }
        if(min_loss > now_loss)
            min_loss = now_loss;
        std::cout << "Round " << i + 1<< ": " << std::endl;
        std::cout << "***************  now_loss  *************" << std::endl;
        std::cout << "***************  " << std::fixed << std::setprecision(10) << now_loss / tot_line_len << "  *************" << std::endl;
        std::cout << "***************  min_loss  *************" << std::endl;
        std::cout << "***************  " << std::fixed << std::setprecision(10) << min_loss / tot_line_len << "  *************" << std::endl;
    }

    cubert_close_tokenizer(tokenizer);
    delete model;
}

void test_train(int batchsize, int seq_length, int nIter, bool base, int num_gpu) {
    bert *model = init_model(base, num_gpu, "", true);

    int test_word_id_seed[11] = {2040, 2001, 3958, 27227, 1029, 3958, 103,
                                 2001, 1037, 13997, 11510};
    int test_token_type_id_seed[11] = {0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1};

    int attention_mask[11] = {1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0};
    int classes[8] = {1, 1, 1, 1, 1, 1, 1, 1};

    int *test_word_id, *test_token_type_id, *test_attention_mask;
    test_word_id = filling_inputs(test_word_id_seed, seq_length, 11, batchsize);
    test_token_type_id = filling_inputs(test_token_type_id_seed, seq_length, 11, batchsize);
    test_attention_mask = filling_inputs(attention_mask, seq_length, 11, batchsize);

    std::cout << "input_ids: " << std::endl;
    for (int i = 0; i < batchsize * seq_length; i++) {
        std::cout << test_word_id[i] << " ";
        if(i % seq_length == seq_length - 1)
            std::cout<< std::endl;
    }
    std::cout << std::endl;

    std::cout << "segment_ids: " << std::endl;
    for (int i = 0; i < batchsize * seq_length; i++) {
        std::cout << test_token_type_id[i] << " ";
        if(i % seq_length == seq_length - 1)
            std::cout<< std::endl;
    }
    std::cout << std::endl;

    std::cout << "input_mask: " << std::endl;
    for (int i = 0; i < batchsize * seq_length; i++) {
        std::cout << test_attention_mask[i] << " ";
        if(i % seq_length == seq_length - 1)
            std::cout<< std::endl;
    }
    std::cout << std::endl;

    std::cout << " Seq_length : " << seq_length << std::endl;
    std::cout << " Batchsize : " << batchsize << std::endl;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *output_pinned;
    checkCudaErrors(hipHostMalloc((void **) &output_pinned,
                                   (1024) * model->handle->hidden_size * sizeof(float)));

    double total_time = 0;
    float learning_rate = 0.001;
    float learning_rate_decay = 0.99;
    for (int i = 0; i < nIter; i++) {
        printf("Round: %d\n", i);
        float it_time;
        hipEventRecord(start);
        model->update_lr_start(learning_rate);
        float loss = cuda_classify_train(
                model,
                test_word_id,
                test_token_type_id,
                classes,
                batchsize,
                seq_length,
                2,
                test_attention_mask
        );
        std::cout << model->handle->learning_rate << std::endl;
        learning_rate *= learning_rate_decay;
        model->update_lr_end();

        printf("loss is %.10f\n", loss);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&it_time, start, stop);
        total_time += it_time;
    }

    delete model;

    double dSeconds = total_time / (double) nIter;
    printf("Time= %.2f(ms)\n", dSeconds);
}
}
