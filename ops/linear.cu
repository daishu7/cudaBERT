#include "hip/hip_runtime.h"
#include "linear.cuh"
#include "elementwise.cuh"
#include "matmul.cuh"

template <typename T> 
void op_Linear::forward (
                        T* &output, 
                        T* input, 
                        size_t n, 
                        size_t k, 
                        size_t m,
                        bool is_prepare,
                        bool debug) {
    output = handle->global_malloc_manage_float.get_new_head_point(n * m);

    if (debug) {
        debug_tensor_gpu<T>(std::string("weights"), kernel, 2, 2, 10);
        debug_tensor_gpu<T>(std::string("bias"), bias, 2, 2, 1);
        debug_tensor_gpu<T>(std::string("input_Linear"), bias, 10, k, min((int)n,10));
    }

    if(!is_prepare){
        dim3 threads(1024, 1, 1);
        dim3 blocks(min((long)65535, n*m/1024) + 1, 1, 1);
        MemoryCpyLinear<T><<<blocks, threads, 0, handle->copy_stream>>>(
                                                 output, bias, n*m, m);
    }
    else{
        checkCudaErrors(hipMemcpyAsync(output,
                                    bias, 
                                    n*m*sizeof(float), 
                                    hipMemcpyDeviceToDevice,
                                    handle->copy_stream));
    }
    
    hipEventRecord(handle->copy_event, handle->copy_stream);

    hipStreamWaitEvent(handle->cal_stream, handle->copy_event, 0);

    if(debug)
        debug_tensor_gpu<T>(std::string("After Linear copy"), output,10, m, min((int)n,10));

    std::vector<size_t> a_shape={n, k};
    std::vector<size_t> b_shape={k, m};
    std::vector<size_t> c_shape={n, m};

    matmul(handle->handle, 
           input, 
           a_shape, 
           kernel, 
           b_shape, 
           output, 
           c_shape,
           false, 
           false, 
           1.0f, 
           1.0f);
    if(debug)
        debug_tensor_gpu<T>(std::string("Linear out"), output, 10, m, min((int)n,10));
}

template 
void op_Linear::forward<float>(
                            float* &output, 
                            float* input, 
                            size_t n, 
                            size_t k, 
                            size_t m,
                            bool is_prepare,
                            bool debug);

template <typename T>
void op_BatchedLinear::forward(
                            T* &output, 
                            T* input, 
                            size_t n, 
                            size_t k, 
                            size_t m,
                            bool is_prepare,
                            bool debug) {
    output = handle->global_malloc_manage_float.get_new_head_point(3 * n * m);
    
    //dim3 threads(512, 1, 1);
    //dim3 blocks(max(3*n*m, 3*k*m)/512 + 1, 1, 1);
    //BatchMemoryCpyLinear<T><<<blocks, threads>>>(weights, output, weights_0, beta_0, weights_1,
    //                            beta_1, weights_2, beta_2, n, k, m);
    if(!is_prepare){
        dim3 threads(1024, 1, 1);
        dim3 blocks(min((long)65535, n*m/1024) + 1, 1, 1);
        MemoryCpyLinear<T><<<blocks, threads, 0, handle->copy_stream>>>(
                                                      output, query_bias, n*m, m);
        MemoryCpyLinear<T><<<blocks, threads, 0, handle->copy_stream>>>(
                                                 output + n*m, key_bias, n*m, m);
        MemoryCpyLinear<T><<<blocks, threads, 0, handle->copy_stream>>>(
                                              output + 2*n*m, val_bias, n*m, m);
    }
    else{
        checkCudaErrors(hipMemcpyAsync(output,
                                    query_bias, 
                                    n*m*sizeof(float), 
                                    hipMemcpyDeviceToDevice,
                                    handle->copy_stream));
        checkCudaErrors(hipMemcpyAsync(output + n*m, 
                                    key_bias, 
                                    n*m*sizeof(float), 
                                    hipMemcpyDeviceToDevice,
                                    handle->copy_stream));
        checkCudaErrors(hipMemcpyAsync(output + 2*n*m, 
                                    val_bias, 
                                    n*m*sizeof(float), 
                                    hipMemcpyDeviceToDevice,
                                    handle->copy_stream));
    }

    //dim3 threads2(512, 1, 1);
    //dim3 blocks2(k*m/512 + 1, 1, 1);
    //MemoryCpyLinear<T><<<blocks2, threads2>>>(weights, weights_0, k*m, k*m);
    //MemoryCpyLinear<T><<<blocks2, threads2>>>(weights + k*m, weights_1, k*m, k*m);
    //MemoryCpyLinear<T><<<blocks2, threads2>>>(weights + 2*k*m, weights_2, k*m, k*m);
    hipEventRecord(handle->copy_event, handle->copy_stream);
    hipStreamWaitEvent(handle->cal_stream, handle->copy_event, 0);

    if(debug){
        //debug_tensor_gpu<T>(std::string("inputs"), input, 10, 768, 11);
        //debug_tensor_gpu<T>(std::string("key"), weights_0, 10, 768, 11);
        //debug_tensor_gpu<T>(std::string("query"), weights_0+k*m, 10, 768, 11);
        //debug_tensor_gpu<T>(std::string("value"), weights_0+2*k*m, 10, 768, 11);
        debug_tensor_gpu<T>(std::string("before matmul"), output, 5, handle->hidden_size*handle->seq_length, handle->batchsize*3);
        //debug_tensor_gpu<T>(std::string("bias"), beta_0, 10, handle->hidden_size, 11);
    }
    
    std::vector<size_t> a_shape={3, n, k};
    std::vector<size_t> b_shape={3, k, m};
    std::vector<size_t> c_shape={3, n, m};

    matmul(handle->handle, 
           input, 
           a_shape, 
           batch_attentin_weights, 
           b_shape, 
           output, 
           c_shape,
           false, 
           false, 
           1.0f, 
           1.0f, 
           0);
    //if(debug)
        //debug_tensor_gpu<T>(std::string("Linear out"), output, 11, 768, 11*3);
}

template
void op_BatchedLinear::forward<float>(
                                    float* &output, 
                                    float* input, 
                                    size_t n, 
                                    size_t k, 
                                    size_t m,
                                    bool is_prepare,
                                    bool debug);