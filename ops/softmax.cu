#include "hip/hip_runtime.h"
#include "softmax.cuh"

template<typename T> __device__ __forceinline__
void cuWelfordMax(
  const T* tensor,
  const int n1,
  const int n2,
  const int i1,
  T& max_) 
{
  max_= T(-99999);
  if (i1 < n1) {
    // one warp normalizes one n1 index,
    // synchronization is implicit
    // initialize with standard Welford algorithm
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const T* lvals = tensor + i1*n2;
    int l = 4*thrx;
    for (;  l+3 < n2;  l+=4*numx) {
      for (int k = 0;  k < 4;  ++k) {
        T curr = static_cast<T>(lvals[l+k]);
        max_ = max(max_, curr);
      }
    }
    for (;  l < n2;  ++l) {
      T curr = static_cast<T>(lvals[l]);
      max_ = max(max_, curr);
    }
    // intra-warp reductions
    for (int l = 0;  l <= 4;  ++l) {
      int srcLaneB = (threadIdx.x+(1<<l))&31;
      T maxB = WARP_SHFL(max_, srcLaneB);
      max_ = max(maxB, max_);
    }
    // threadIdx.x == 0 has correct values for each warp
    // inter-warp reductions{
    max_ = WARP_SHFL(max_, 0);
  }
}

template<typename T> __device__ __forceinline__
void cuWelfordSum(
  const T* vals,
  const int n1,
  const int n2,
  const int i1,
  T& sum) 
{
  sum = T(0);
  if (i1 < n1) {
    // one warp normalizes one n1 index,
    // synchronization is implicit
    // initialize with standard Welford algorithm
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const T* lvals = vals + i1*n2;
    int l = 4*thrx;
    for (;  l+3 < n2;  l+=4*numx) {
      for (int k = 0;  k < 4;  ++k) {
        T curr = static_cast<T>(lvals[l+k]);
        sum += curr;
      }
    }
    for (;  l < n2;  ++l) {
      T curr = static_cast<T>(lvals[l]);
      sum += curr;
    }
    // intra-warp reductions
    for (int l = 0;  l <= 4;  ++l) {
      int srcLaneB = (threadIdx.x+(1<<l))&31;
      T sumB = WARP_SHFL(sum, srcLaneB);
      sum += sumB;
    }
    // threadIdx.x == 0 has correct values for each warp
    // inter-warp reductions{
    sum = WARP_SHFL(sum, 0);
  }
}


template<typename T> __global__
void cuApplySoftmax(
  T* tensor,
  const int n1,
  const int n2
  ) 
{
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensors are contiguous
  //
  for(int i1 = blockIdx.y; i1 < n1; i1 += gridDim.y){
    T max_, sum;
    cuWelfordMax(tensor,n1,n2,i1,max_);
    T* vals = tensor + i1*n2;
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    for (int i = thrx;  i < n2;  i+=numx) {
        vals[i] = exp(vals[i] - max_);
    }
    cuWelfordSum(tensor,n1,n2,i1,sum);
    for (int i = thrx;  i < n2;  i+=numx) {
        vals[i] = vals[i] / sum;
    }
    __syncthreads();
  }
}

template<typename T> 
void HostApplySoftmax(
    global_manager *handle,
    T* tensor,
    size_t n1,
    size_t n2
    )
{
    const dim3 threads(32,1,1);
    const dim3 blocks(1,min((long)65535,n1),1);
    cuApplySoftmax<<<blocks, threads, 0, handle->get_cal_stream()>>>(
		    tensor,
		    n1,n2
            );
}

template 
void HostApplySoftmax<float>(
    global_manager *handle,
    float* tensor,
    size_t n1,
    size_t n2
    );
