#include "hip/hip_runtime.h"
#include "layernorm.cuh"

#include "shfl.cuh"
#include "../utils/common.h"
#include "../utils/manager.cuh"

template<typename U> __device__ U rsqrt(U v) {
  return U(1) / sqrt(v);
}

template<typename U> __device__
void cuWelfordOnlineSum(
  const U curr,
  U& mu,
  U& sigma2,
  U& count)
{
  count = count + U(1);
  U delta = curr - mu;
  U lmean = mu + delta / count;
  mu = lmean;
  U delta2 = curr - lmean;
  sigma2 = sigma2 + delta * delta2;
}


template<typename U> __device__
void cuChanOnlineSum(
  const U muB,
  const U sigma2B,
  const U countB,
  U& mu,
  U& sigma2,
  U& count)
{
  U delta = muB - mu;
  U nA = count;
  U nB = countB;
  count = count + countB;
  U nX = count;
  if (nX > U(0)) {
    nA = nA / nX;
    nB = nB / nX;
    mu = nA*mu + nB*muB;
    sigma2 = sigma2 + sigma2B + delta * delta * nA * nB * nX;
  } else {
    mu = U(0);
    sigma2 = U(0);
  }
}

template<typename T, typename U> __device__
void cuWelfordMuSigma2(
  const T* __restrict__ vals,
  const int n1,
  const int n2,
  const int i1,
  U& mu,
  U& sigma2,
  U* buf) 
{
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensor is contiguous
  // 3) 2*blockDim.y*sizeof(U)+blockDim.y*sizeof(int) shared memory available.
  //
  // compute variance and mean over n2
  U count = U(0);
  mu= U(0);
  sigma2 = U(0);
  if (i1 < n1) {
    // one warp normalizes one n1 index,
    // synchronization is implicit
    // initialize with standard Welford algorithm
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const T* lvals = vals + i1*n2;
    int l = 4*thrx;
    for (;  l+3 < n2;  l+=4*numx) {
      for (int k = 0;  k < 4;  ++k) {
        U curr = static_cast<U>(lvals[l+k]);
        cuWelfordOnlineSum<U>(curr,mu,sigma2,count);
      }
    }
    for (;  l < n2;  ++l) {
      U curr = static_cast<U>(lvals[l]);
      cuWelfordOnlineSum<U>(curr,mu,sigma2,count);
    }
    // intra-warp reductions
    for (int l = 0;  l <= 4;  ++l) {
      int srcLaneB = (threadIdx.x+(1<<l))&31;
      U muB = WARP_SHFL(mu, srcLaneB);
      U countB = WARP_SHFL(count, srcLaneB);
      U sigma2B = WARP_SHFL(sigma2, srcLaneB);
      cuChanOnlineSum<U>(muB,sigma2B,countB,mu,sigma2,count);
    }
    // threadIdx.x == 0 has correct values for each warp
    // inter-warp reductions
    if (blockDim.y > 1) {
      U* ubuf = (U*)buf;
      U* ibuf = (U*)(ubuf + blockDim.y);
      for (int offset = blockDim.y/2;  offset > 0;  offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.x == 0 && threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int wrt_y = threadIdx.y - offset;
          ubuf[2*wrt_y] = mu;
          ubuf[2*wrt_y+1] = sigma2;
          ibuf[wrt_y] = count;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.x == 0 && threadIdx.y < offset) {
          U muB = ubuf[2*threadIdx.y];
          U sigma2B = ubuf[2*threadIdx.y+1];
          U countB = ibuf[threadIdx.y];
          cuChanOnlineSum<U>(muB,sigma2B,countB,mu,sigma2,count);
        }
        __syncthreads();
      }
      // threadIdx.x = 0 && threadIdx.y == 0 only thread that has correct values
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        ubuf[0] = mu;
        ubuf[1] = sigma2;
      }
      __syncthreads();
      mu = ubuf[0];
      sigma2 = ubuf[1]/U(n2);
      // don't care about final value of count, we know count == n2
    } else {
      mu = WARP_SHFL(mu, 0);
      sigma2 = WARP_SHFL(sigma2/U(n2), 0);
    }
  }
}

template<typename T, typename U> __global__
void cuApplyLayerNorm(
  T* __restrict__ output_vals,
  T* __restrict__ vals,
  const int n1,
  const int n2,
  const U epsilon,
  const T* __restrict__ gamma,
  const T* __restrict__ beta,
  const T* merge_add
  ) 
{
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensors are contiguous
  //
  if(merge_add != nullptr){
    for(int i1 = blockIdx.y; i1 < n1; i1 += gridDim.y){
      for(int i = threadIdx.y * blockDim.x  + threadIdx.x ; i < n2; i += blockDim.y * blockDim.x)
        vals[i + i1 * n2] += merge_add[i + i1 * n2];
    }
    __syncthreads();
  }

  for(int i1 = blockIdx.y; i1 < n1; i1 += gridDim.y){
    SharedMemory<U> shared;
    U* buf = shared.getPointer();
    U mu,sigma2;
    cuWelfordMuSigma2(vals,n1,n2,i1,mu,sigma2,buf);
    const T* lvals = vals + i1*n2;
    T* ovals = output_vals + i1*n2;
    U c_invvar = rsqrt<U>(sigma2 + epsilon);
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    if (gamma != NULL && beta != NULL) {
      for (int i = thrx;  i < n2;  i+=numx) {
        U curr = static_cast<U>(lvals[i]);
        ovals[i] = gamma[i] * static_cast<T>(c_invvar * (curr - mu)) + beta[i];
      }
    } else {
      for (int i = thrx;  i < n2;  i+=numx) {
        U curr = static_cast<U>(lvals[i]);
        ovals[i] = static_cast<T>(c_invvar * (curr - mu));
      }
    }
  }
}

template<typename T> 
void op_LayerNorm::forward(
    T* output,
    T* input,
    size_t n1,
    size_t n2,
    T* merge_add
    )
{
    // auto stream TODO(): Muti-Stream 
    const dim3 threads(32,4,1);
    const dim3 blocks(1,min((long)65535,n1),1);
    int nshared = 
        threads.y > 1 ? 
	    threads.y*sizeof(T)+(threads.y/2)*sizeof(T) : 
	    0;
    cuApplyLayerNorm<<<blocks, threads, nshared, handle->cal_stream>>>(
		    output,
		    input,
		    n1,n2,
		    T(epsilon),
          gamma,beta, merge_add);
}

template
void op_LayerNorm::forward<float>(
    float* output,
    float* input,
    size_t n1,
    size_t n2,
    float* merge_add
    );
