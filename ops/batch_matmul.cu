#include "hip/hip_runtime.h"
#include "batch_matmul.cuh"

void blas_sgemm_batch(  hipblasHandle_t handle,
                        const bool TransA, const bool TransB,
                        int m, int n, int k,
                        const float alpha,
                        const float **Aarray, int lda,
                        const float **Barray, int ldb,
                        const float beta,
                        float **Carray, int ldc,
                        int batchCount) {
    checkCudaErrors(hipblasSgemmBatched(handle,
                                    TransA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                                    TransB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                                    m, n, k,
                                    &alpha,
                                    Aarray, lda,
                                    Barray, ldb,
                                    &beta,
                                    Carray, ldc,
                                    batchCount));                 
}

__global__ void load_pointer_vector_qk(const float* query,
                                        const float* key,
                                        float* out,
                                        const float** query_array,
                                        const float** key_array,
                                        float** out_array,
                                        size_t batchsize,
                                        size_t seq_length,
                                        size_t num_attention_heads,
                                        size_t length_per_heads){
    size_t id_batchsize = threadIdx.x;
    size_t id_heads = blockIdx.x;
    size_t idx = id_batchsize * seq_length * length_per_heads * num_attention_heads +
                                length_per_heads * id_heads;
    query_array[id_batchsize * num_attention_heads + id_heads] = query + idx;
    key_array[id_batchsize * num_attention_heads + id_heads] = key + idx;
    out_array[id_batchsize * num_attention_heads + id_heads] = out +
        id_batchsize * seq_length * seq_length * num_attention_heads + id_heads * seq_length;

    __syncthreads();
}



void Query_Key::forward(const float* query,
                        const float* key,
                        float* &output){
    size_t batchsize = handle->batchsize;
    size_t seq_length = handle->seq_length;
    size_t num_attention_heads = handle->num_attention_heads;
    size_t length_per_heads = handle->hidden_size / num_attention_heads;

    output =  handle->global_malloc_manage_float.get_new_head_point(
                        batchsize * num_attention_heads * seq_length * seq_length);
    
    dim3 threads(batchsize, 1, 1);
    dim3 blocks(num_attention_heads, 1, 1);
    load_pointer_vector_qk<<<blocks, threads, 0, handle->cal_stream>>>(query,
                                                                       key,
                                                                       output,
                                                                       query_array,
                                                                       key_array,
                                                                       out_array,
                                                                       batchsize,
                                                                       seq_length,
                                                                       num_attention_heads,
                                                                       length_per_heads);
    
    blas_sgemm_batch(handle->handle,
                    true, false,
                    seq_length, seq_length, length_per_heads,
                    1.0,
                    key_array, num_attention_heads * length_per_heads,
                    query_array, num_attention_heads * length_per_heads,
                    0.0,
                    out_array, num_attention_heads * seq_length,
                    num_attention_heads * batchsize);
    
}

__global__ void load_pointer_vector_pv(const float* prob,
                                        const float* value,
                                        float* out,
                                        const float** prob_array,
                                        const float** value_array,
                                        float** out_array,
                                        size_t batchsize,
                                        size_t seq_length,
                                        size_t num_attention_heads,
                                        size_t length_per_heads){
    size_t id_batchsize = threadIdx.x;
    size_t id_heads = blockIdx.x;
    size_t idx = id_batchsize * seq_length * length_per_heads * num_attention_heads +
                                length_per_heads * id_heads;
    prob_array[id_batchsize * num_attention_heads + id_heads] = prob +         
         id_batchsize * seq_length * seq_length * num_attention_heads + id_heads * seq_length;
    value_array[id_batchsize * num_attention_heads + id_heads] = value + idx;
    out_array[id_batchsize * num_attention_heads + id_heads] = out + idx;

    __syncthreads();
}

void Prob_Value::forward(const float* prob,
                        const float* value,
                        float* &output){
    size_t batchsize = handle->batchsize;
    size_t seq_length = handle->seq_length;
    size_t num_attention_heads = handle->num_attention_heads;
    size_t length_per_heads = handle->hidden_size / num_attention_heads;

    output =  handle->global_malloc_manage_float.get_new_head_point(
                        batchsize * num_attention_heads * seq_length * seq_length);
    
    dim3 threads(batchsize, 1, 1);
    dim3 blocks(num_attention_heads, 1, 1);
    load_pointer_vector_pv<<<blocks, threads, 0, handle->cal_stream>>>(prob,
                                                                       value,
                                                                       output,
                                                                       prob_array,
                                                                       value_array,
                                                                       out_array,
                                                                       batchsize,
                                                                       seq_length,
                                                                       num_attention_heads,
                                                                       length_per_heads);
    
    blas_sgemm_batch(handle->handle,
                    false, false,
                    length_per_heads, seq_length, seq_length,
                    1.0,
                    value_array, num_attention_heads * length_per_heads,
                    prob_array, num_attention_heads * seq_length,
                    0.0,
                    out_array, num_attention_heads * length_per_heads,
                    num_attention_heads * batchsize);
    
}