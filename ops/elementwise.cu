#include "hip/hip_runtime.h"
#include "elementwise.cuh"

#include "math.h"

template <typename T> 
__global__ void MemoryCpyLinear(T* out, T* in, size_t max, size_t warpsize) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x ; i < max; i += gridDim.x * blockDim.x)
        out[i] = in[i%warpsize];
    __syncthreads();
}

template <typename T>
__global__ void device_copy_pooler(T* out, 
                                   T* in, 
                                   size_t hidden_size, 
                                   size_t seq_length,
                                   size_t batchsize){
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x ; 
           i < batchsize * hidden_size; i += gridDim.x * blockDim.x){
        size_t num_batch = i / hidden_size;
        out[i] = in[num_batch * seq_length * hidden_size + i%hidden_size];
    }
    __syncthreads();
}

template <typename T>
void copy_pooler(T* &output, T* tensor, global_handle* handle){
    output = handle->global_malloc_manage_float.get_new_head_point(
                            handle->batchsize * handle->hidden_size);

    dim3 threads(handle->hidden_size, 1, 1);
    dim3 blocks(min(long(65535), handle->batchsize), 1, 1);
    device_copy_pooler<<<blocks, threads, 0, handle->cal_stream>>>(
                                    output,
                                    tensor,
                                    handle->hidden_size,
                                    handle->seq_length,
                                    handle->batchsize);
}

template 
void copy_pooler<float>(float* &output, float* tensor, global_handle* handle);

template
__global__ void MemoryCpyLinear<float>(float* out, float* in, size_t max, size_t warpsize);

template <typename T> 
__device__ void BertTranspose (T* out, 
                              T *in, 
                              size_t batchsize, 
                              size_t seq_length, 
                              size_t num_heads, 
                              long total_length,
                              bool muti_head) {
  for(size_t index = blockIdx.x; index < total_length; index += gridDim.x){
    size_t remain = blockDim.x / num_heads;
    size_t size_head = seq_length * remain;
    size_t num_batch = index / seq_length;
    size_t size_batch = blockDim.x * seq_length;
    size_t num_head = threadIdx.x / remain;
    size_t num_seq = index % seq_length;
    size_t size_seq = remain;
    size_t forward_index = num_batch*size_batch + num_head*size_head + num_seq*size_seq + threadIdx.x%remain;
    size_t backward_index = blockDim.x * index + threadIdx.x;
    if(muti_head){    
        out[forward_index] = in[backward_index];
    }
    else{
        out[backward_index] = in[forward_index];
    }
  }
  __syncthreads();
}

template <typename T>
__global__ void FusionTranspose (T* out,
                                T* in, 
                                int num, 
                                size_t batchsize, 
                                size_t seq_length,
                                size_t num_heads,
                                size_t total_length,
                                size_t total, 
                                bool muti_head) {
    for(int i = 0; i < num; i++)
        BertTranspose(out + i*total, in + i*total, batchsize, seq_length, num_heads, total_length, muti_head);
    __syncthreads();
}

template <typename T>
void op_FusionTranspose::forward (T* &out,
                                T* in, 
                                int num, 
                                bool muti_head) {
    out = handle->global_malloc_manage_float.get_new_head_point(
            handle->batchsize * handle->seq_length * handle->hidden_size * 3);
    dim3 threads(handle->hidden_size, 1, 1);
    dim3 blocks(min(long(65535), handle->batchsize * handle->seq_length), 1, 1);

    FusionTranspose<<<blocks, threads, 0, handle->cal_stream>>>(
                        out,
                        in,
                        num, 
                        handle->batchsize, 
                        handle->seq_length,
                        handle->num_attention_heads,
                        handle->batchsize * handle->seq_length,
                        handle->batchsize * handle->seq_length * handle->hidden_size, 
                        muti_head);
}

template
void op_FusionTranspose::forward<float>(float* &out,
                                        float* in, 
                                        int num, 
                                        bool muti_head);

template <typename T>
__global__ void mask (
                    T* tensor, 
                    int* mask, 
                    float number, 
                    size_t max_num, 
                    size_t batchsize, 
                    size_t seq_length) {
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x ; i < max_num; i += gridDim.x * blockDim.x){
        size_t index = seq_length * ( i / ( max_num / batchsize )) + i % seq_length;
        if(mask != nullptr)
            tensor[i] = tensor[i]/number + (1 - mask[index]) * -10000.0;
        else
            tensor[i] = tensor[i]/number;
    } 
    __syncthreads();
}

template <typename T>
void op_Mask_Add::forward (
                    T* tensor, 
                    int* attention_mask, 
                    float number) {
    size_t seq_length = handle->seq_length;
    size_t batchsize = handle->batchsize;
    size_t num_attention_heads = handle->num_attention_heads;

    dim3 threads(1024, 1, 1);
    dim3 blocks(min( (long)65535, 
            seq_length*seq_length*batchsize*num_attention_heads / 1024) + 1, 1, 1);
    mask<<<blocks, threads, 0, handle->cal_stream>>>(
                  tensor, 
                  attention_mask, 
                  number, 
                  batchsize * seq_length * num_attention_heads * seq_length, 
                  batchsize, 
                  seq_length); 
}

template
void op_Mask_Add::forward<float>(
                            float* tensor, 
                            int* mask, 
                            float number);

template <typename T>
__global__ void gelu (T* tensor,  size_t max_num) {
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x ; i < max_num; i += gridDim.x * blockDim.x)
        tensor[i] = tensor[i] * 0.5f * (1.0f + erff(tensor[i] / sqrtf(2.0)));
    __syncthreads();
}

template <typename T>
void op_Gelu::forward (T* tensor, size_t max_num) {
    dim3 threads(1024, 1, 1);
    dim3 blocks(min((long)65535, max_num / 1024) + 1, 1, 1);
    gelu<<<blocks, threads, 0, handle->cal_stream>>>(
                                     tensor, 
                                     max_num); 
}

template
void op_Gelu::forward<float>(float* tensor, size_t max_num);

template <typename T>
__global__ void Tanh (T* tensor,  size_t max_num) {
    for(size_t i = blockIdx.x * blockDim.x + threadIdx.x ; i < max_num; i += gridDim.x * blockDim.x)
        tensor[i] = tanh(tensor[i]);
    __syncthreads();
}

template <typename T>
void op_Tanh::forward (T* tensor, size_t max_num){
    dim3 threads(1024, 1, 1);
    dim3 blocks(min((long)65535, max_num / 1024) + 1, 1, 1);
    Tanh<<<blocks, threads, 0, handle->cal_stream>>>(
                                    tensor, 
                                    max_num);
}

template
void op_Tanh::forward<float>(float* tensor,  size_t max_num);
