#include "hip/hip_runtime.h"
#ifndef LINEAR_CUDA_BERT
#define LINEAR_CUDA_BERT

// CUDA and CUBLAS functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "matmul.cu"
#include "../utils/manager.h"

template <typename T>
__global__ void MemoryCpyLinear(T* out, T* in, int max, int warpsize) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x ; i < max; i += gridDim.x * blockDim.x)
        out[i] = in[i%warpsize];
    __syncthreads();
}

template <typename T>
__global__ void BatchMemoryCpyLinear (
        T* weights_out, 
        T* beta_out, 
        T* weights_0, 
        T* beta_0,
        T* weights_1, 
        T* beta_1, 
        T* weights_2, 
        T* beta_2, 
        size_t n, 
        size_t k, 
        size_t m) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x ; i < 3 * n * m; i += gridDim.x * blockDim.x) {
        size_t num_beta = i / (n * m);
        switch(num_beta){
            case 0 : {beta_out[i] = beta_0[i % m]; break;}
            case 1 : {beta_out[i] = beta_1[i % m]; break;}
            case 2 : {beta_out[i] = beta_2[i % m]; break;}
        }
    }
    for (int i = blockIdx.x * blockDim.x + threadIdx.x ; i < 3 * k * m; i += gridDim.x * blockDim.x) {
        size_t num_weights = i / (k * m);
        switch (num_weights) {
            case 0 : {weights_out[i] = weights_0[i % (k * m)]; break;}
            case 1 : {weights_out[i] = weights_1[i % (k * m)]; break;}
            case 2 : {weights_out[i] = weights_2[i % (k * m)]; break;}
        }
    }
    __syncthreads();
}

template <typename T> 
void Linear (global_manager *handler, 
        T* output, 
        T* input, 
        T* weights, 
        T* beta,
        size_t n, 
        size_t k, 
        size_t m,
        bool is_prepare = false,
        bool debug=false) {

    if (debug) {
        debug_tensor_gpu<T>(std::string("weights"), weights, 2, 2, 10);
        debug_tensor_gpu<T>(std::string("bias"), beta, 2, 2, 1);
        debug_tensor_gpu<T>(std::string("input_Linear"), input, 10, k, min((int)n,10));
    }

    if(!is_prepare){
        dim3 threads(1024, 1, 1);
        dim3 blocks(min((long)65535, n*m/1024) + 1, 1, 1);
        MemoryCpyLinear<T><<<blocks, threads, 0, handler->get_copy_stream()>>>(
                                                 output, beta, n*m, m);
    }
    else{
        checkCudaErrors(hipMemcpyAsync(output,
                                    beta, 
                                    n*m*sizeof(float), 
                                    hipMemcpyDeviceToDevice,
                                    handler->get_copy_stream()));
    }
    
    hipEventRecord(handler->copy_event, handler->get_copy_stream());

    hipStreamWaitEvent(handler->get_cal_stream(), handler->copy_event, 0);

    if(debug)
        debug_tensor_gpu<T>(std::string("After Linear copy"), output,10, m, min((int)n,10));

    std::vector<size_t> a_shape={n, k};
    std::vector<size_t> b_shape={k, m};
    std::vector<size_t> c_shape={n, m};

    matmul(handler->handle, 
           input, 
           a_shape, 
           weights, 
           b_shape, 
           output, 
           c_shape,
           false, 
           false, 
           1.0f, 
           1.0f);
    if(debug)
        debug_tensor_gpu<T>(std::string("Linear out"), output, 10, m, min((int)n,10));
}

template <typename T>
void Batch_Linear (global_manager *handler, 
                   T* output, 
                   T* input, 
                   T* weights_0, 
                   T* beta_0,
                   T* weights_1, 
                   T* beta_1, 
                   T* weights_2, 
                   T* beta_2, 
                   size_t n, 
                   size_t k, 
                   size_t m,
                   bool is_prepare = false,
                   bool debug = false) {
    T* weights = handler->global_malloc_manage_float.get_new_head_point(3 * k * m);
    
    //dim3 threads(512, 1, 1);
    //dim3 blocks(max(3*n*m, 3*k*m)/512 + 1, 1, 1);
    //BatchMemoryCpyLinear<T><<<blocks, threads>>>(weights, output, weights_0, beta_0, weights_1,
    //                            beta_1, weights_2, beta_2, n, k, m);
    if(!is_prepare){
        dim3 threads(1024, 1, 1);
        dim3 blocks(min((long)65535, n*m/1024) + 1, 1, 1);
        MemoryCpyLinear<T><<<blocks, threads, 0, handler->get_copy_stream()>>>(
                                                      output, beta_0, n*m, m);
        MemoryCpyLinear<T><<<blocks, threads, 0, handler->get_copy_stream()>>>(
                                                 output + n*m, beta_1, n*m, m);
        MemoryCpyLinear<T><<<blocks, threads, 0, handler->get_copy_stream()>>>(
                                              output + 2*n*m, beta_2, n*m, m);
    }
    else{
        checkCudaErrors(hipMemcpyAsync(output,
                                    beta_0, 
                                    n*m*sizeof(float), 
                                    hipMemcpyDeviceToDevice,
                                    handler->get_copy_stream()));
        checkCudaErrors(hipMemcpyAsync(output + n*m, 
                                    beta_1, 
                                    n*m*sizeof(float), 
                                    hipMemcpyDeviceToDevice,
                                    handler->get_copy_stream()));
        checkCudaErrors(hipMemcpyAsync(output + 2*n*m, 
                                    beta_2, 
                                    n*m*sizeof(float), 
                                    hipMemcpyDeviceToDevice,
                                    handler->get_copy_stream()));
    }

    dim3 threads2(512, 1, 1);
    dim3 blocks2(k*m/512 + 1, 1, 1);
    //MemoryCpyLinear<T><<<blocks2, threads2>>>(weights, weights_0, k*m, k*m);
    //MemoryCpyLinear<T><<<blocks2, threads2>>>(weights + k*m, weights_1, k*m, k*m);
    //MemoryCpyLinear<T><<<blocks2, threads2>>>(weights + 2*k*m, weights_2, k*m, k*m);
    hipEventRecord(handler->copy_event, handler->get_copy_stream());
    hipStreamWaitEvent(handler->get_cal_stream(), handler->copy_event, 0);

    if(debug){
        //debug_tensor_gpu<T>(std::string("inputs"), input, 10, 768, 11);
        //debug_tensor_gpu<T>(std::string("key"), weights_0, 10, 768, 11);
        //debug_tensor_gpu<T>(std::string("query"), weights_0+k*m, 10, 768, 11);
        //debug_tensor_gpu<T>(std::string("value"), weights_0+2*k*m, 10, 768, 11);
        debug_tensor_gpu<T>(std::string("before matmul"), output, 5, handler->hidden_size*handler->seq_length, handler->batchsize*3);
        //debug_tensor_gpu<T>(std::string("bias"), beta_0, 10, handler->hidden_size, 11);
    }
    
    std::vector<size_t> a_shape={3, n, k};
    std::vector<size_t> b_shape={3, k, m};
    std::vector<size_t> c_shape={3, n, m};

    matmul(handler->handle, 
           input, 
           a_shape, 
           weights_0, 
           b_shape, 
           output, 
           c_shape,
           false, 
           false, 
           1.0f, 
           1.0f, 
           0);
    //if(debug)
        //debug_tensor_gpu<T>(std::string("Linear out"), output, 11, 768, 11*3);
}

#endif
