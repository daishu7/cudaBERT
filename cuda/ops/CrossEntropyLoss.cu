#include "hip/hip_runtime.h"
#ifndef CROSSENTROPYLOSS_CUDA_BERT
#define CROSSENTROPYLOSS_CUDA_BERT

#include "../utils/common.h"
#include "../utils/manager.h"
#include "softmax.cu"

template <typename T, typename U>
__global__ void cuApplyCrossEntropyLoss(
    T *__restrict__ output_vals, T *__restrict__ input, U *__restrict__ classes, const int n1, const int n2, T *__restrict__ weight = nullptr)
{
    for (int i1 = blockIdx.y; i1 < n1; i1 += gridDim.y)
    {
        T sum, max_;
        cuWelfordMax(input, n1, n2, i1, max_);
        T *vals = input + i1 * n2;
        const int blockx = blockDim.x * blockDim.y;
        const int threadx = threadIdx.x + threadIdx.y * blockDim.x;
        for (int i = threadx; i < n2; i += blockx)
        {
            vals[i] = exp(vals[i] - max_);
        }
        cuWelfordSum(input, n1, n2, i1, sum);
        const int label = static_cast<int>(classes[i1]);
        if (weight != nullptr)
        {
            output_vals[i1] = weight[label] * (-log(input[n2 * i1 + label]) + log(sum));
        }
        else
        {
            output_vals[i1] = -log(input[n2 * i1 + label]) + log(sum);
        }
        __syncthreads();
    }
}

template <typename T>
__global__ void cuApplyCrossEntropyLossAverage(T *output, const int n1, const int n2)
{
    T average_output = 0;
    for (int index = blockIdx.x; index < n1; index += gridDim.x)
    {
        average_output += output[index];
    }
    if (blockIdx.x == 0)
        output[n1] = average_output / n1;
}

template <typename T, typename U>
void HostApplyCrossEntropyLoss(
    global_manager *handle, T *output, T *input, U *classes,
    size_t n1, size_t n2, T *weight = nullptr)
{
    const dim3 threads(32, 1, 1);
    const dim3 blocks(1, min((long)65535, n1), 1);
    cuApplyCrossEntropyLoss<<<blocks, threads, 0, handle->get_cal_stream()>>>(
        output, input, classes, n1, n2, weight);

    const dim3 threads1(min((long)32, n1), 1, 1);
    const dim3 blocks1(1, 1, 1);
    cuApplyCrossEntropyLossAverage<<<blocks1, threads1, 0, handle->get_cal_stream()>>>(
        output, n1, n2);
}

template <typename T, typename U>
__global__ void cuApplyCrossEntropyLossGradient(
    T *__restrict__ dout, T *__restrict__ input, U *__restrict__ classes, const int n1, const int n2, T *grad_input, T *__restrict__ weight = nullptr)
{
    for (int i1 = blockIdx.y; i1 < n1; i1 += gridDim.y)
    {
        T sum, max_;
        cuWelfordMax(input, n1, n2, i1, max_);
        T *vals = input + i1 * n2;
        const int numx = blockDim.x * blockDim.y;
        const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
        for (int i = thrx; i < n2; i += numx)
        {
            vals[i] = exp(vals[i] - max_);
        }
        cuWelfordSum(input, n1, n2, i1, sum);
        const int label = static_cast<int>(classes[i1]);
        T *k_grad_input = grad_input + i1 * n2;
        if (weight != nullptr)
        {
            for (int l = thrx; l < n2; l += numx)
            {
                T f_grad_input = 1.0 / n1 * vals[l] / sum * weight[label];
                if (l == label)
                {
                    f_grad_input -= 1.0 / n1 * weight[label];
                    k_grad_input[l] = f_grad_input * dout[i1];
                }
                else
                {
                    k_grad_input[l] = f_grad_input * dout[i1];
                }
            }
        }
        else
        {
            for (int l = thrx; l < n2; l += numx)
            {
                T f_grad_input = 1.0 / n1 * vals[l] / sum;
                if (l == label)
                {
                    f_grad_input -= 1.0 / n1;
                    k_grad_input[l] = f_grad_input * dout[i1];
                }
                else
                {
                    k_grad_input[l] = f_grad_input * dout[i1];
                }
            }
        }
        __syncthreads();
    }
}

template <typename T, typename U>
void HostApplyCrossEntropyLossGradient(
    global_manager *handle,
    T *dout,
    T *input,
    size_t n1, size_t n2, U *classes,
    T *grad_input, T *weight = nullptr)
{
    const dim3 threads(32, 1, 1);
    const dim3 blocks(1, min((long)65535, n1), 1);
    cuApplyCrossEntropyLossGradient<<<blocks, threads, 0, handle->get_cal_stream()>>>(
        dout, input, classes, n1, n2, grad_input, weight);
    debug_tensor_gpu<float>(std::string("Grid CrossEntropyLoss_output"), grad_input, 2, 2, n1);
}
#endif